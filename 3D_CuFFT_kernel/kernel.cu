#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>   
#include <stdlib.h>  
#include <math.h>  
#include <hip/hip_runtime.h> 
#include <hipfft/hipfft.h> 
#include <memory.h> 
#include<time.h>
#pragma comment( lib, "cufft.lib" )

#define N1 128 
#define N2 128
#define N3 16

#define CN3 ((int)N3/2+1) // half N3

void functf(double A[2][N1][N2][N3]);

void FFT3d_GPU(double A[2][N1][N2][N3]);

void main(void)
{

	static double A[2][N1][N2][N3];
	functf(A);

	FFT3d_GPU(A);
	printf("OK! \n");
}

void FFT3d_GPU(double A[2][N1][N2][N3])
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	const int    FFT_SIZE_R = N1 * N2 * N3  * sizeof(hipfftDoubleReal);

	const int    FFT_SIZE_C = N1 * N2 * CN3 * sizeof(hipfftDoubleComplex);

	hipfftDoubleReal *h_oRealData = (hipfftDoubleReal*)malloc(FFT_SIZE_R); 
	memset(h_oRealData, 0x00, FFT_SIZE_R);
	hipfftDoubleComplex *h_otestComplexData = (hipfftDoubleComplex*)malloc(FFT_SIZE_C);
	memset(h_otestComplexData, 0x00, FFT_SIZE_C);

	hipfftDoubleReal *d_iRealData;
	hipMalloc((void**)&d_iRealData, FFT_SIZE_R);
	hipMemcpy(d_iRealData, A[0], FFT_SIZE_R, hipMemcpyHostToDevice);

	hipfftDoubleComplex *d_oComplexData;
	hipMalloc((void**)&d_oComplexData, FFT_SIZE_C);

	hipfftDoubleReal *d_oRealData;
	hipMalloc((void**)&d_oRealData, FFT_SIZE_R);
	// /* Create a 3D FFT plan for D2Z */
	printf("cufft FFT,direct transform(x-->A), A:\n");
	hipEventRecord(start, 0);
	hipfftHandle planD2Z3D;
	hipfftPlan3d(&planD2Z3D, N1, N2, N3, HIPFFT_D2Z);
	hipfftExecD2Z(planD2Z3D, (hipfftDoubleReal*)d_iRealData, d_oComplexData);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Kernel time: %.2f ms\n", elapsedTime);

	hipMemcpy(h_otestComplexData, d_oComplexData, FFT_SIZE_C, hipMemcpyDeviceToHost);
	hipEventRecord(start, 0);
	//Create a 3D FFT plan for Z2D. 
	hipfftHandle  planZ2D3D;
	hipfftPlan3d(&planZ2D3D, N1, N2, N3, HIPFFT_Z2D);
	//D2Z out of  place
	printf("\n\nAfter 3D C2R out of place : \n");
	printf("cuFFT IFFT,inverse transform(x-->A), A:\n");
	// Use the CUFFT plan to transform the signal out of place.  
	hipfftExecZ2D(planZ2D3D, d_oComplexData, (hipfftDoubleReal*)d_oRealData);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime1;
	hipEventElapsedTime(&elapsedTime1, start, stop);
	printf("iCufft Kernel time: %.2f ms\n", elapsedTime1);
	printf("total Kernel time: %.2f ms\n", elapsedTime1 + elapsedTime);
	hipMemcpy(h_oRealData, d_oRealData, FFT_SIZE_R, hipMemcpyDeviceToHost);

//	printf("Total running time: %f seconds\n", (double)(totaltime1) / CLOCKS_PER_SEC);
	unsigned int i, j, k;
	FILE *fp;
	fp = fopen("fft3_cuda.d", "w");
	//system("cls");

	// origianl data
	printf("file copy\n");
	fprintf(fp, "the original data, AK:\n");
	for (k = 0; k<N3; k++)
		for (j = 0; j<N2; j++)
			for (i = 0; i<N1; i++)
				fprintf(fp, "%15.9f\n", A[0][i][j][k]);
	fprintf(fp, "\n");

	// direct transform
	fprintf(fp, "cufft FFT,direct transform(x-->A), A:\n");
	for (k = 0; k<CN3; k++)
		for (j = 0; j<N2; j++)
			for (i = 0; i<N1; i++)
				fprintf(fp, "%4u,%4u,%4u:%15.9f,%16.8e\n", i, j, k, h_otestComplexData[i*N2*CN3 + j*CN3 + k].x, h_otestComplexData[i*N2*CN3 + j*CN3 + k].y);
	fprintf(fp, "\n");

	//inverse transform

	fprintf(fp, "cu FFT IFFT,inverse transform(x-->A), A:\n");
	for (k = 0; k<N3; k++)
		for (j = 0; j<N2; j++)
			for (i = 0; i<N1; i++)
				fprintf(fp, "%4u,%4u,%4u:%15.9f\n", i, j, k, h_oRealData[i*N2*N3 + j*N3 + k] / (N1*N2*N3));
	fprintf(fp, "\n");

}
/*****************************************************/
void functf(double A[2][N1][N2][N3])
{
	unsigned int i, j, k;
	double tti, ttj, ttk, dti, dtj, dtk, A0i, A0j, A0k, A1i, A1j, A1k;
	dti = 0.1;  dtj = 2.0*dti;  dtk = dtj;
	A0i = exp(-N1*dti);
	A0j = exp(-N2*dtj);
	A0k = exp(-N3*dtk);

	for (i = 0; i<N1; i++)
	{
		tti = (double)i*dti;
		A1i = exp(-tti);
		for (j = 0; j<N2; j++)
		{
			ttj = (double)j*dtj;
			A1j = exp(-ttj);
			for (k = 0; k<N3; k++)
			{
				ttk = (double)k*dtk;
				A1k = exp(-ttk);

				A[0][i][j][k] = (A1i + A0i / A1i)*dti*(A1j + A0j / A1j)*dtj*(A1k + A0k / A1k)*dtk;
				A[1][i][j][k] = 0.0;
			}
		}
	}
}
